#include "hip/hip_runtime.h"
//ƒ==============================================================================================
// Originally written in 2016 by Peter Shirley <ptrshrl@gmail.com>
//
// To the extent possible under law, the author(s) have dedicated all copyright and related and
// neighboring rights to this software to the public domain worldwide. This software is
// distributed without any warranty.
//
// You should have received a copy (see file COPYING.txt) of the CC0 Public Domain Dedication
// along with this software. If not, see <http://creativecommons.org/publicdomain/zero/1.0/>.
//==============================================================================================

#include "rtweekend.h"
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include "camera.h"
#include "hittable.h"
#include <time.h>
#include "hittable_list.h"
#include "material.h"
#include "sphere.h"

// typedef hipError_t hipError_t;

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

long material_size(material* mat) {
    if (metal* d = dynamic_cast<metal*>(mat)) {
        return sizeof(metal);
    } 
    else if (lambertian* d = dynamic_cast<lambertian*>(mat)) {
        return sizeof(lambertian);
    } 
    else if (dielectric* d = dynamic_cast<dielectric*>(mat)) {
        return sizeof(dielectric);
    }
    return sizeof(*mat);
}

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void create_world(hittable_list* world, sphere* sphere_list) {
    // printf("According to the list, the first object is at %p\n", &sphere_list[1]);
    for (int i =0; i < world->size(); i++){
        world->objects[i] = &sphere_list[i];
        // printf("%c\n", ((world->objects[i])->mat)->type);
    }
}

__global__ void render_init(camera *cam, hiprandState *rand_state) {
    int i = (blockIdx.x * blockDim.x + threadIdx.x);
    int j = (blockIdx.y * blockDim.y + threadIdx.y);
    int pixel_index = i + (*cam).image_width*j;

    // printf("%d\n", pixel_index);

    if (i >= (*cam).image_width || j >= (*cam).image_height)
        return;

    hiprand_init(1984 + pixel_index, 0, 0, &rand_state[pixel_index]);
    // printf("Done");
}

// Code that will be run on the GPU
__global__ void render(const hittable_list *world, camera *cam, hiprandState *rand_state, color *pixel_color) {
    int i = (blockIdx.x * blockDim.x + threadIdx.x);
    int j = (blockIdx.y * blockDim.y + threadIdx.y);
    int pixel_index = i + (*cam).image_width*j;

    if (i >= (*cam).image_width || j >= (*cam).image_height)
        return;

    hiprandState local_rand_state = rand_state[pixel_index];
    
    // ((hittable_list *)world)->balls();
    
    for (int sample = 0; sample < (*cam).samples_per_pixel; sample++) {
        ray r = (*cam).get_ray(i, j, &local_rand_state);
        // printf("Post Malone \n");
        // printf("%d %p\n", (*cam).max_depth, &local_rand_state);
        pixel_color[pixel_index] += (*cam).ray_color(r, (*cam).max_depth, world, &local_rand_state);
        // printf("%f %f %f\n", pixel_color[pixel_index].x(), pixel_color[pixel_index].y(), pixel_color[pixel_index].z());
    }
}


int main(int argc, char **argv) {
    if (argc < 3){
        printf("Format : ./main imageWidth numThreadsPerBlock_x numThreadsPerBlock_y");
    }

    int numThreadsPerBlock_x = atoi(argv[2]);
    int numThreadsPerBlock_y = atoi(argv[3]);

    hittable_list world;

    //auto ground_material = make_shared<lambertian>(color(0.5, 0.5, 0.5));
    lambertian* ground_material = new lambertian(color(0.5, 0.5, 0.5)); // Replacing the above case.
    //world.add(make_shared<sphere>(point3(0,-1000,0), 1000, ground_material));
    world.add(new sphere(point3(0,-1000,0), 1000, ground_material));    // sphere is the pointer type

    for (int a = -11; a < 11; a++) {
        for (int b = -11; b < 11; b++) {
            auto choose_mat = std::rand() / (RAND_MAX + 1.0);
            point3 center(a + 0.9*std::rand() / (RAND_MAX + 1.0), 0.2, b + 0.9*std::rand() / (RAND_MAX + 1.0));

            if ((center - point3(4, 0.2, 0)).length() > 0.9) {
                material* sphere_material;

                if (choose_mat < 0.8) {
                    // diffuse
                    auto albedo = color::random() * color::random();
                    sphere_material = new lambertian(albedo);
                    world.add(new sphere(center, 0.2, sphere_material));
                } else if (choose_mat < 0.95) {
                    // metal
                    auto albedo = color::random(0.5, 1);
                    auto fuzz = 0.5 * std::rand() / (RAND_MAX + 1.0);
                    sphere_material = new metal(albedo, fuzz);
                    world.add(new sphere(center, 0.2, sphere_material));
                } else {
                    // glass
                    sphere_material = new dielectric(1.5);
                    world.add(new sphere(center, 0.2, sphere_material));
                }
            }
        }
    }

    auto material1 = new dielectric(1.5);
    world.add(new sphere(point3(0, 1, 0), 1.0, material1));

    auto material2 = new lambertian(color(0.4, 0.2, 0.1));
    world.add(new sphere(point3(-4, 1, 0), 1.0, material2));

    auto material3 = new metal(color(0.7, 0.6, 0.5), 0.0);
    world.add(new sphere(point3(4, 1, 0), 1.0, material3));

    camera cam;

    cam.aspect_ratio      = 16.0 / 9.0;
    cam.image_width       = atoi(argv[1]);
    cam.samples_per_pixel = 10;
    cam.max_depth         = 2;

    cam.vfov     = 20;
    cam.lookfrom = point3(13,2,3);
    cam.lookat   = point3(0,0,0);
    cam.vup      = vec3(0,1,0);

    cam.defocus_angle = 0.6;
    cam.focus_dist    = 10.0;
    // printf("world.size() = %d\n", world.size());
    // printf("camera's image width = %d\n", cam.image_width);
    // printf("Number of threads per block = %d, %d\n", numThreadsPerBlock_x, numThreadsPerBlock_y);
    
    clock_t start, stop;
    start = clock();
    cam.initialize();                                               // Need to call this once before we pass things to the GPU
    
    // CUDA doesn't have std::rand() therefore we need to define a set of seds
    void* d_rand_state;         // Abse we will start using pointers as just that -- pointers. We don't think of pointers of being of a specifc class
    hipMalloc((void **)&d_rand_state, cam.image_height * cam.image_height * sizeof(hiprandState));
    
    void* pixel_color;                                  // Making it a pointer for the sake of my sanity.
    hipMalloc((void **)&pixel_color, cam.image_width * cam.image_height * sizeof(color));        // We will allocate memory in the GPU for storing the pixel colors
    
    // Allocating appropriate space in GPU
    int num_entries = world.size();
    
    void* d_cam;                // Making it a pointer for the sake of my sanity.
    hipMalloc((void **)&d_cam, sizeof(camera));
    // Copying relevant data
    hipMemcpy(d_cam, (void *)&cam, sizeof(camera), hipMemcpyHostToDevice);

    void* d_list_of_spheres; // It is just a pointer for now. It points to the memory location which contiguously contains all of the objects
    hipMalloc((void **)&d_list_of_spheres, world.size() * sizeof(sphere));
    checkCudaErrors(hipGetLastError());

    // Sharing every sphere with the GPU. Note that we also need to send the material to GPU
    void* material_ptr;
    std::cout << "Size of lambertian = " << sizeof(lambertian) << std::endl;
    std::cout << "Size of dielectric = " << sizeof(dielectric) << std::endl;
    std::cout << "Size of metal = " << sizeof(metal) << std::endl;
    std::cout << "Size of material = " << sizeof(material) << std::endl;

    for (int i = 0; i < world.size(); i ++){        
        // Let us first allocate space for the material that makes the sphere.
        hipMalloc((void **)&material_ptr, material_size((*world.objects[i]).mat));
        // Let us now copy the material
        hipMemcpy(material_ptr, (*world.objects[i]).mat, material_size((*world.objects[i]).mat), hipMemcpyHostToDevice);
        // Let us update the sphere's pointer to material before we send it over
        world.objects[i]->mat = (material *)material_ptr;
        // Sending the material now fr
        hipMemcpy((void *)&(((sphere *)d_list_of_spheres)[i]), (void *)world.objects[i], sizeof(sphere), hipMemcpyHostToDevice);
        // Let us now update the pointer to the sphere in the world
        world.objects[i] = &(((sphere *)d_list_of_spheres)[i]);
    }
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Now we will share the object of class hittable_list with the GPU
    void* d_world;
    hipMalloc((void **)&d_world, sizeof(hittable_list));
    hipMemcpy(d_world, &world, sizeof(hittable_list), hipMemcpyHostToDevice);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    // Created a world on the GPU and we even have the location of that as a pointer

    std::cout << "No problems yet\n";
    // This will update the world object in the GPU to accurately point to the actual spheres
    create_world<<<1, 1>>>((hittable_list *)d_world, (sphere *)d_list_of_spheres);         // The sole job of this is to create the world using the pointer to the list and the update the d_world pointer accordingly
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    dim3 blocks(cam.image_height/numThreadsPerBlock_x + 1,cam.image_width/numThreadsPerBlock_y + 1);
    dim3 threads(numThreadsPerBlock_x, numThreadsPerBlock_y);
    
    render_init<<<blocks, threads>>>((camera *)d_cam, (hiprandState *)d_rand_state);     // We are passing pointers to the camera object, rand_state and world -- all of these are already in the GPU memory
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render<<<blocks, threads>>>((hittable_list *)d_world, 
                                (camera *)d_cam, 
                                (hiprandState *)d_rand_state, 
                                (color *)pixel_color);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();

    // printf("Rnedingnineign\n");

    color* h_pixel_color = (color*)malloc(cam.image_width * cam.image_height * sizeof(color));
    hipMemcpy(h_pixel_color, pixel_color, cam.image_width * cam.image_height * sizeof(color), hipMemcpyDeviceToHost);
    for (int j = 0 ; j < cam.image_height; j++){
        for (int i = 0 ; i < cam.image_width; i++){
            write_color(std::cout, cam.pixel_samples_scale * h_pixel_color[i + cam.image_width*j]);
        }
    }

    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << timer_seconds << "\n";

}
